
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS  1000000
#define MINPOINTS 20

static void handleError(hipError_t err, const char *file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR(err) (handleError(err, __FILE__, __LINE__))

void checkParam();
__global__ void initLine(float*, float*, int);
__global__ void updateAll(float*, float*, int, int);
void printResult();

int totalSteps, totalPoints, allocPoints;
float *currVal;
float *devCurrVal, *devPrevVal;

int main(int argc, char *argv[]) {
	sscanf(argv[1], "%d", &totalPoints);
	sscanf(argv[2], "%d", &totalSteps);
	checkParam();

	allocPoints = totalPoints + 256;

	currVal = (float*) malloc(allocPoints * sizeof(float));
	if (!currVal)
		exit(EXIT_FAILURE);

	HANDLE_ERROR(hipMalloc((void**) &devCurrVal, allocPoints * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**) &devPrevVal, allocPoints * sizeof(float)));

	dim3 threadsPerBlock(256);
	dim3 numOfBlocks(allocPoints/256);

	printf("Initializing points on the line...\n");
	initLine<<<numOfBlocks, threadsPerBlock>>>(devPrevVal, devCurrVal, totalPoints);

	printf("Updating all points for all time steps...\n");
	updateAll<<<numOfBlocks, threadsPerBlock>>>(devPrevVal, devCurrVal, totalPoints, totalSteps);

	printf("Printing final results...\n");
	HANDLE_ERROR(hipMemcpy(currVal, devCurrVal, allocPoints * sizeof(float), hipMemcpyDeviceToHost));
	printResult();

	printf("\nDone.\n\n");

	hipFree(devCurrVal);
	hipFree(devPrevVal);

	free(currVal);

	return EXIT_SUCCESS;
}

void checkParam() {
	char temp[20];
	while ((totalPoints < MINPOINTS) || (totalPoints > MAXPOINTS)) {
		printf("Enter number of points along vibrating string [%d-%d]: ", MINPOINTS, MAXPOINTS);
		scanf("%s", temp);
		totalPoints = atoi(temp);
		if ((totalPoints < MINPOINTS) || (totalPoints > MAXPOINTS))
			printf("Invalid. Please enter value between %d and %d.\n", MINPOINTS, MAXPOINTS);
	}
	while ((totalSteps < 1) || (totalSteps > MAXSTEPS)) {
		printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
		scanf("%s", temp);
		totalSteps = atoi(temp);
		if ((totalSteps < 1) || (totalSteps > MAXSTEPS))
			printf("Invalid. Please enter value between 1 and %d.\n", MAXSTEPS);
	}
	printf("Using points = %d, steps = %d\n", totalPoints, totalSteps);
}

__global__ void initLine(float *__devPrevVal, float *__devCurrVal, int __totalPoints) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < __totalPoints) {
		float x = (float) i / (__totalPoints - 1);
		__devPrevVal[i] = __devCurrVal[i] = __sinf(6.28318530 * x);
	}
}

__global__ void updateAll(float *__devPrevVal, float *__devCurrVal, int __totalPoints, int __totalSteps) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < __totalPoints) {
		float locPrevVal = __devPrevVal[i], locCurrVal = __devCurrVal[i] , locNextVal;
		for (int j = 0; j < __totalSteps; j++) {
			if ((i == 0) || (i == __totalPoints - 1))
				locNextVal = 0.0;
			else
				locNextVal = 1.82 * locCurrVal - locPrevVal;
			locPrevVal = locCurrVal;
			locCurrVal = locNextVal;
		}
		__devCurrVal[i] = locCurrVal;
	}
}

void printResult() {
	for (int i = 0; i < totalPoints; i++) {
		printf("%6.4f ", currVal[i]);
		if ((i + 1) % 10 == 0)
			printf("\n");
	}
}